
#include <hip/hip_runtime.h>
#include<iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

__global__ void kernel( float* r_gpu,  float* g_gpu,   float* b_gpu, int N) {
	
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if(tId < N) {
		r_gpu[tId] = 1 - r_gpu[tId];
		g_gpu[tId] = 1 - g_gpu[tId];
		b_gpu[tId] = 1 - b_gpu[tId];
	}
}



void CambiarColores(float* r, float* g, float* b) {
	//cout << *r << "|" << 1-*r << endl;
	*r = 1 - *r;
	*g = 1 - *g;
	*b = 1 - *b;
}


int main(int argc, char const *argv[]) {
	FILE * pFile;
	int n, m;
	float* r, *g, *b; 



	pFile = fopen ("img.txt","r");
	fscanf(pFile, "%d %d", &m, &n);


	int block_size = 256;
	int grid_size = (int) ceil((float) n*m / block_size);


	float* r_gpu, *g_gpu, *b_gpu;

	hipMalloc(&r_gpu, sizeof(float) * n * m);
	hipMalloc(&g_gpu, sizeof(float) * n * m);
	hipMalloc(&b_gpu, sizeof(float) * n * m);

	r = new float[n*m];
	g = new float[n*m];
	b = new float[n*m];

	for (int i = 0; i < n*m; ++i) {
		fscanf (pFile, "%f", &r[i]);
	}

	for (int i = 0; i < n*m; ++i) {
		fscanf (pFile, "%f", &g[i]);
	}

	for (int i = 0; i < n*m; ++i) {
		fscanf (pFile, "%f", &b[i]);
	}

	fclose (pFile);

	hipMemcpy(r_gpu, r, sizeof(float) * n * m, hipMemcpyHostToDevice);
	hipMemcpy(g_gpu, g, sizeof(float) * n * m, hipMemcpyHostToDevice);
	hipMemcpy(b_gpu, b, sizeof(float) * n * m, hipMemcpyHostToDevice);

	int tamanio = n * m;

	hipEvent_t ct1, ct2;
	float dt;
	hipEventCreate(&ct1);
	hipEventCreate(&ct2);
	hipEventRecord(ct1);

	kernel<<<grid_size, block_size>>>(r_gpu, g_gpu, b_gpu, tamanio);

	hipEventRecord(ct2);
	hipEventSynchronize(ct2);
	hipEventElapsedTime(&dt, ct1, ct2);

	cout << "Tiempo GPU: " << dt << " [ms]" << endl; 

	hipMemcpy(r, r_gpu, sizeof(float) * n * m, hipMemcpyDeviceToHost);
	hipMemcpy(g, g_gpu, sizeof(float) * n * m, hipMemcpyDeviceToHost);
	hipMemcpy(b, b_gpu, sizeof(float) * n * m, hipMemcpyDeviceToHost);


	hipFree(r_gpu);
	hipFree(g_gpu);
	hipFree(b_gpu);






	FILE * pSalida;
	pSalida = fopen ("img_salida.txt","w");
	fprintf(pSalida, "%d %d\n", m, n);
	for (int i = 0; i < n*m; ++i) {
		if(i == n*m - 1) {
			fprintf(pSalida, "%f", r[i]);
		} else {
			fprintf(pSalida, "%f ", r[i]);
		}
		
	}
	fprintf(pSalida, "\n");
	for (int i = 0; i < n*m; ++i) {
		if(i == n*m - 1) {
			fprintf(pSalida, "%f", g[i]);
		} else {
			fprintf(pSalida, "%f ", g[i]);
		}
	}
	fprintf(pSalida, "\n");
	for (int i = 0; i < n*m; ++i) {
		if(i == n*m - 1) {
			fprintf(pSalida, "%f", b[i]);
		} else {
			fprintf(pSalida, "%f ", b[i]);
		}
	}
	
	delete r;
	delete g;
	delete b;
	//cin.get();
	return 0;
}